#include "hip/hip_runtime.h"
// REQUIRES: nvptx-registered-target

// Make sure we don't allow dynamic initialization for device
// variables, but accept empty constructors allowed by CUDA.

// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fcuda-is-device -std=c++11 \
// RUN:     -fno-threadsafe-statics -emit-llvm -o - %s | FileCheck %s
// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fcuda-is-device -std=c++11 \
// RUN:     -emit-llvm -DERROR_CASE -verify -o /dev/null %s

#ifdef __clang__
#include "Inputs/hip/hip_runtime.h"
#endif

// Base classes with different initializer variants.

// trivial constructor -- allowed
struct T {
  int t;
};

// empty constructor
struct EC {
  int ec;
  __device__ EC() {}     // -- allowed
  __device__ EC(int) {}  // -- not allowed
};

// empty templated constructor -- allowed with no arguments
struct ETC {
  template <typename... T> __device__ ETC(T...) {}
};

// undefined constructor -- not allowed
struct UC {
  int uc;
  __device__ UC();
};

// empty constructor w/ initializer list -- not allowed
struct ECI {
  int eci;
  __device__ ECI() : eci(1) {}
};

// non-empty constructor -- not allowed
struct NEC {
  int nec;
  __device__ NEC() { nec = 1; }
};

// no-constructor,  virtual method -- not allowed
struct NCV {
  int ncv;
  __device__ virtual void vm() {}
};

// dynamic in-class field initializer -- not allowed
__device__ int f();
struct NCF {
  int ncf = f();
};

// static in-class field initializer.  NVCC does not allow it, but
// clang generates static initializer for this, so we'll accept it.
struct NCFS {
  int ncfs = 3;
};

// undefined templated constructor -- not allowed
struct UTC {
  template <typename... T> __device__ UTC(T...);
};

// non-empty templated constructor -- not allowed
struct NETC {
  int netc;
  template <typename... T> __device__ NETC(T...) { netc = 1; }
};

__device__ int d_v;
// CHECK: @d_v = addrspace(1) externally_initialized global i32 0,
__shared__ int s_v;
// CHECK: @s_v = addrspace(3) global i32 undef,
__constant__ int c_v;
// CHECK: addrspace(4) externally_initialized global i32 0,

__device__ int d_v_i = 1;
// CHECK: @d_v_i = addrspace(1) externally_initialized global i32 1,
#ifdef ERROR_CASE
__shared__ int s_v_i = 1;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
#endif
__constant__ int c_v_i = 1;
// CHECK: @c_v_i = addrspace(4) externally_initialized global i32 1,

#ifdef ERROR_CASE
__device__ int d_v_f = f();
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ int s_v_f = f();
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ int c_v_f = f();
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
#endif

__device__ T d_t;
// CHECK: @d_t = addrspace(1) externally_initialized global %struct.T zeroinitializer
__shared__ T s_t;
// CHECK: @s_t = addrspace(3) global %struct.T undef,
__constant__ T c_t;
// CHECK: @c_t = addrspace(4) externally_initialized global %struct.T zeroinitializer,

__device__ T d_t_i = {2};
// CHECKL @d_t_i = addrspace(1) externally_initialized global %struct.T { i32 2 },
#ifdef ERROR_CASE
__shared__ T s_t_i = {2};
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
#endif
__constant__ T c_t_i = {2};
// CHECK: @c_t_i = addrspace(4) externally_initialized global %struct.T { i32 2 },

__device__ EC d_ec;
// CHECK: @d_ec = addrspace(1) externally_initialized global %struct.EC zeroinitializer,
__shared__ EC s_ec;
// CHECK: @s_ec = addrspace(3) global %struct.EC undef,
__constant__ EC c_ec;
// CHECK: @c_ec = addrspace(4) externally_initialized global %struct.EC zeroinitializer,

#if ERROR_CASE
__device__ EC d_ec_i(3);
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ EC s_ec_i(3);
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ EC c_ec_i(3);
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ EC d_ec_i2 = {3};
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ EC s_ec_i2 = {3};
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ EC c_ec_i2 = {3};
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
#endif

__device__ ETC d_etc;
// CHETCK: @d_etc = addrspace(1) externally_initialized global %struct.ETC zeroinitializer,
__shared__ ETC s_etc;
// CHETCK: @s_etc = addrspace(3) global %struct.ETC undef,
__constant__ ETC c_etc;
// CHETCK: @c_etc = addrspace(4) externally_initialized global %struct.ETC zeroinitializer,

#if ERROR_CASE
__device__ ETC d_etc_i(3);
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ ETC s_etc_i(3);
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ ETC c_etc_i(3);
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ ETC d_etc_i2 = {3};
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ ETC s_etc_i2 = {3};
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ ETC c_etc_i2 = {3};
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ UC d_uc;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ UC s_uc;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ UC c_uc;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ ECI d_eci;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ ECI s_eci;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ ECI c_eci;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ NEC d_nec;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ NEC s_nec;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ NEC c_nec;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ NCV d_ncv;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ NCV s_ncv;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ NCV c_ncv;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ NCF d_ncf;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ NCF s_ncf;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ NCF c_ncf;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
#endif

__device__ NCFS d_ncfs;
// CHECK: @d_ncfs = addrspace(1) externally_initialized global %struct.NCFS { i32 3 }
__constant__ NCFS c_ncfs;
// CHECK: @c_ncfs = addrspace(4) externally_initialized global %struct.NCFS { i32 3 }

#if ERROR_CASE
__shared__ NCFS s_ncfs;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}

__device__ UTC d_utc;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ UTC s_utc;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ UTC c_utc;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ UTC d_utc_i(3);
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ UTC s_utc_i(3);
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ UTC c_utc_i(3);
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ NETC d_netc;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ NETC s_netc;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ NETC c_netc;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ NETC d_netc_i(3);
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ NETC s_netc_i(3);
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ NETC c_netc_i(3);
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
#endif

// Regular base class -- allowed
struct T_B_T : T {};
__device__ T_B_T d_t_b_t;
// CHECK: @d_t_b_t = addrspace(1) externally_initialized global %struct.T_B_T zeroinitializer,
__shared__ T_B_T s_t_b_t;
// CHECK: @s_t_b_t = addrspace(3) global %struct.T_B_T undef,
__constant__ T_B_T c_t_b_t;
// CHECK: @c_t_b_t = addrspace(4) externally_initialized global %struct.T_B_T zeroinitializer,

// Incapsulated object of allowed class -- allowed
struct T_F_T {
  T t;
};
__device__ T_F_T d_t_f_t;
// CHECK: @d_t_f_t = addrspace(1) externally_initialized global %struct.T_F_T zeroinitializer,
__shared__ T_F_T s_t_f_t;
// CHECK: @s_t_f_t = addrspace(3) global %struct.T_F_T undef,
__constant__ T_F_T c_t_f_t;
// CHECK: @c_t_f_t = addrspace(4) externally_initialized global %struct.T_F_T zeroinitializer,

// array of allowed objects -- allowed
struct T_FA_T {
  T t[2];
};
__device__ T_FA_T d_t_fa_t;
// CHECK: @d_t_fa_t = addrspace(1) externally_initialized global %struct.T_FA_T zeroinitializer,
__shared__ T_FA_T s_t_fa_t;
// CHECK: @s_t_fa_t = addrspace(3) global %struct.T_FA_T undef,
__constant__ T_FA_T c_t_fa_t;
// CHECK: @c_t_fa_t = addrspace(4) externally_initialized global %struct.T_FA_T zeroinitializer,


// Calling empty base class initializer is OK
struct EC_I_EC : EC {
  __device__ EC_I_EC() : EC() {}
};
__device__ EC_I_EC d_ec_i_ec;
// CHECK: @d_ec_i_ec = addrspace(1) externally_initialized global %struct.EC_I_EC zeroinitializer,
__shared__ EC_I_EC s_ec_i_ec;
// CHECK: @s_ec_i_ec = addrspace(3) global %struct.EC_I_EC undef,
__constant__ EC_I_EC c_ec_i_ec;
// CHECK: @c_ec_i_ec = addrspace(4) externally_initialized global %struct.EC_I_EC zeroinitializer,

// .. though passing arguments is not allowed.
struct EC_I_EC1 : EC {
  __device__ EC_I_EC1() : EC(1) {}
};
#if ERROR_CASE
__device__ EC_I_EC1 d_ec_i_ec1;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ EC_I_EC1 s_ec_i_ec1;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ EC_I_EC1 c_ec_i_ec1;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
#endif

// Virtual base class -- not allowed
struct T_V_T : virtual T {};
#if ERROR_CASE
__device__ T_V_T d_t_v_t;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ T_V_T s_t_v_t;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ T_V_T c_t_v_t;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
#endif

// Make sure that we don't allow if we inherit or incapsulate
// something with disallowed initializer.

// Inherited from or incapsulated class with non-empty constructor --
// not allowed
struct T_B_NEC : NEC {};
struct T_F_NEC {
  NEC nec;
};
struct T_FA_NEC {
  NEC nec[2];
};

#if ERROR_CASE
__device__ T_B_NEC d_t_b_nec;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ T_B_NEC s_t_b_nec;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ T_B_NEC c_t_b_nec;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ T_F_NEC d_t_f_nec;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ T_F_NEC s_t_f_nec;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ T_F_NEC c_t_f_nec;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ T_FA_NEC d_t_fa_nec;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ T_FA_NEC s_t_fa_nec;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ T_FA_NEC c_t_fa_nec;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
#endif

// We should not emit global initializers for device-side variables.
// CHECK-NOT: @__cxx_global_var_init

// Make sure that initialization restrictions do not apply to local
// variables.
__device__ void df() {
  T t;
  EC ec;
  ETC etc;
  UC uc;
  ECI eci;
  NEC nec;
  NCV ncv;
  NCF ncf;
  NCFS ncfs;
  UTC utc;
  NETC netc;
  T_B_T t_b_t;
  T_F_T t_f_t;
  T_FA_T t_fa_t;
  EC_I_EC ec_i_ec;
  EC_I_EC1 ec_i_ec1;
  T_V_T t_v_t;
  T_B_NEC t_b_nec;
  T_F_NEC t_f_nec;
  T_FA_NEC t_fa_nec;
  static __shared__ UC s_uc;
}

// CHECK:   call void @_ZN2ECC1Ev(%struct.EC* %ec)
// CHECK:   call void @_ZN3ETCC1IJEEEDpT_(%struct.ETC* %etc)
// CHECK:   call void @_ZN2UCC1Ev(%struct.UC* %uc)
// CHECK:   call void @_ZN3ECIC1Ev(%struct.ECI* %eci)
// CHECK:   call void @_ZN3NECC1Ev(%struct.NEC* %nec)
// CHECK:   call void @_ZN3NCVC1Ev(%struct.NCV* %ncv)
// CHECK:   call void @_ZN3NCFC1Ev(%struct.NCF* %ncf)
// CHECK:   call void @_ZN4NCFSC1Ev(%struct.NCFS* %ncfs)
// CHECK:   call void @_ZN3UTCC1IJEEEDpT_(%struct.UTC* %utc)
// CHECK:   call void @_ZN4NETCC1IJEEEDpT_(%struct.NETC* %netc)
// CHECK:   call void @_ZN7EC_I_ECC1Ev(%struct.EC_I_EC* %ec_i_ec)
// CHECK:   call void @_ZN8EC_I_EC1C1Ev(%struct.EC_I_EC1* %ec_i_ec1)
// CHECK:   call void @_ZN5T_V_TC1Ev(%struct.T_V_T* %t_v_t) #3
// CHECK:   call void @_ZN7T_B_NECC1Ev(%struct.T_B_NEC* %t_b_nec)
// CHECK:   call void @_ZN7T_F_NECC1Ev(%struct.T_F_NEC* %t_f_nec)
// CHECK:   call void @_ZN8T_FA_NECC1Ev(%struct.T_FA_NEC* %t_fa_nec)
// CHECK:   call void @_ZN2UCC1Ev(%struct.UC* addrspacecast (%struct.UC addrspace(3)* @_ZZ2dfvE4s_uc to %struct.UC*))
// CHECK: ret void

// We should not emit global init function.
// CHECK-NOT: @_GLOBAL__sub_I
