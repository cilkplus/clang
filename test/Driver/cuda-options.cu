
#include <hip/hip_runtime.h>
// Tests CUDA compilation pipeline construction in Driver.
// REQUIRES: clang-driver
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// Simple compilation case:
// RUN: %clang -### -target x86_64-linux-gnu -c %s 2>&1 \
// Compile device-side to PTX assembly and make sure we use it on the host side.
// RUN:   | FileCheck -check-prefix CUDA-D1 -check-prefix CUDA-D1NS\
// Then compile host side and incorporate device code.
// RUN:   -check-prefix CUDA-H -check-prefix CUDA-H-I1 \
// Make sure we don't link anything.
// RUN:   -check-prefix CUDA-NL %s

// Typical compilation + link case:
// RUN: %clang -### -target x86_64-linux-gnu %s 2>&1 \
// Compile device-side to PTX assembly and make sure we use it on the host side
// RUN:   | FileCheck -check-prefix CUDA-D1 -check-prefix CUDA-D1NS\
// Then compile host side and incorporate device code.
// RUN:   -check-prefix CUDA-H -check-prefix CUDA-H-I1 \
// Then link things.
// RUN:   -check-prefix CUDA-L %s

// Verify that --cuda-host-only disables device-side compilation and linking
// RUN: %clang -### -target x86_64-linux-gnu --cuda-host-only %s 2>&1 \
// Make sure we didn't run device-side compilation.
// RUN:   | FileCheck -check-prefix CUDA-ND \
// Then compile host side and make sure we don't attempt to incorporate GPU code.
// RUN:    -check-prefix CUDA-H -check-prefix CUDA-H-NI \
// Linking is allowed to happen, even if we're missing GPU code.
// RUN:    -check-prefix CUDA-L %s

// Same test as above, but with preceeding --cuda-device-only to make
// sure only last option has effect.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only --cuda-host-only %s 2>&1 \
// Make sure we didn't run device-side compilation.
// RUN:   | FileCheck -check-prefix CUDA-ND \
// Then compile host side and make sure we don't attempt to incorporate GPU code.
// RUN:    -check-prefix CUDA-H -check-prefix CUDA-H-NI \
// Linking is allowed to happen, even if we're missing GPU code.
// RUN:    -check-prefix CUDA-L %s

// Verify that --cuda-device-only disables host-side compilation and linking
// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only %s 2>&1 \
// Compile device-side to PTX assembly
// RUN:   | FileCheck -check-prefix CUDA-D1 -check-prefix CUDA-D1NS\
// Make sure there are no host cmpilation or linking.
// RUN:   -check-prefix CUDA-NH -check-prefix CUDA-NL %s

// Same test as above, but with preceeding --cuda-host-only to make
// sure only last option has effect.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-host-only --cuda-device-only %s 2>&1 \
// Compile device-side to PTX assembly
// RUN:   | FileCheck -check-prefix CUDA-D1 -check-prefix CUDA-D1NS\
// Make sure there are no host cmpilation or linking.
// RUN:   -check-prefix CUDA-NH -check-prefix CUDA-NL %s

// Verify that with -S we compile host and device sides to assembly
// and incorporate device code on the host side.
// RUN: %clang -### -target x86_64-linux-gnu -S -c %s 2>&1 \
// Compile device-side to PTX assembly
// RUN:   | FileCheck -check-prefix CUDA-D1 -check-prefix CUDA-D1NS\
// Then compile host side and incorporate GPU code.
// RUN:  -check-prefix CUDA-H -check-prefix CUDA-H-I1 \
// Make sure we don't link anything.
// RUN:  -check-prefix CUDA-NL %s

// Verify that --cuda-gpu-arch option passes correct GPU
// archtecture info to device compilation.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-gpu-arch=sm_35 -c %s 2>&1 \
// Compile device-side to PTX assembly.
// RUN:   | FileCheck -check-prefix CUDA-D1 -check-prefix CUDA-D1NS \
// RUN:   -check-prefix CUDA-D1-SM35 \
// Then compile host side and incorporate GPU code.
// RUN:   -check-prefix CUDA-H -check-prefix CUDA-H-I1 \
// Make sure we don't link anything.
// RUN:   -check-prefix CUDA-NL %s

// Verify that there is device-side compilation per --cuda-gpu-arch args
// and that all results are included on the host side.
// RUN: %clang -### -target x86_64-linux-gnu \
// RUN:        --cuda-gpu-arch=sm_35 --cuda-gpu-arch=sm_30 -c %s 2>&1 \
// Compile both device-sides to PTX assembly
// RUN:   | FileCheck \
// RUN: -check-prefix CUDA-D1 -check-prefix CUDA-D1NS -check-prefix CUDA-D1-SM35 \
// RUN: -check-prefix CUDA-D2 -check-prefix CUDA-D2-SM30 \
// Then compile host side and incorporate both device-side outputs
// RUN:   -check-prefix CUDA-H -check-prefix CUDA-HNS \
// RUN:   -check-prefix CUDA-H-I1 -check-prefix CUDA-H-I2 \
// Make sure we don't link anything.
// RUN:   -check-prefix CUDA-NL %s

// Verify that device-side results are passed to correct tool when
// -save-temps is used
// RUN: %clang -### -target x86_64-linux-gnu -save-temps -c %s 2>&1 \
// Compile device-side to PTX assembly and make sure we use it on the host side.
// RUN:   | FileCheck -check-prefix CUDA-D1 -check-prefix CUDA-D1S \
// Then compile host side and incorporate device code.
// RUN:   -check-prefix CUDA-H -check-prefix CUDA-HS -check-prefix CUDA-HS-I1 \
// Make sure we don't link anything.
// RUN:   -check-prefix CUDA-NL %s

// Verify that device-side results are passed to correct tool when
// -fno-integrated-as is used
// RUN: %clang -### -target x86_64-linux-gnu -fno-integrated-as -c %s 2>&1 \
// Compile device-side to PTX assembly and make sure we use it on the host side.
// RUN:   | FileCheck -check-prefix CUDA-D1 -check-prefix CUDA-D1NS \
// Then compile host side and incorporate device code.
// RUN:   -check-prefix CUDA-H -check-prefix CUDA-HNS -check-prefix CUDA-HS-I1 \
// RUN:   -check-prefix CUDA-H-AS \
// Make sure we don't link anything.
// RUN:   -check-prefix CUDA-NL %s

// --cuda-host-only should never trigger unused arg warning.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-host-only -c %s 2>&1 | \
// RUN:    FileCheck -check-prefix CUDA-NO-UNUSED-CHO %s
// RUN: %clang -### -target x86_64-linux-gnu --cuda-host-only -x c -c %s 2>&1 | \
// RUN:    FileCheck -check-prefix CUDA-NO-UNUSED-CHO %s

// --cuda-device-only should not produce warning compiling CUDA files
// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only -c %s 2>&1 | \
// RUN:    FileCheck -check-prefix CUDA-NO-UNUSED-CDO %s

// --cuda-device-only should warn during non-CUDA compilation.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only -x c -c %s 2>&1 | \
// RUN:    FileCheck -check-prefix CUDA-UNUSED-CDO %s

// Match device-side preprocessor, and compiler phases with -save-temps
// CUDA-D1S: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// CUDA-D1S-SAME: "-aux-triple" "x86_64--linux-gnu"
// CUDA-D1S-SAME: "-fcuda-is-device"
// CUDA-D1S-SAME: "-x" "cuda"

// CUDA-D1S: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// CUDA-D1S-SAME: "-aux-triple" "x86_64--linux-gnu"
// CUDA-D1S-SAME: "-fcuda-is-device"
// CUDA-D1S-SAME: "-x" "cuda-cpp-output"

// Match the job that produces PTX assembly
// CUDA-D1: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// CUDA-D1NS-SAME: "-aux-triple" "x86_64--linux-gnu"
// CUDA-D1-SAME: "-fcuda-is-device"
// CUDA-D1-SM35-SAME: "-target-cpu" "sm_35"
// CUDA-D1-SAME: "-o" "[[GPUBINARY1:[^"]*]]"
// CUDA-D1NS-SAME: "-x" "cuda"
// CUDA-D1S-SAME: "-x" "ir"

// Match another device-side compilation
// CUDA-D2: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// CUDA-D2-SAME: "-aux-triple" "x86_64--linux-gnu"
// CUDA-D2-SAME: "-fcuda-is-device"
// CUDA-D2-SM30-SAME: "-target-cpu" "sm_30"
// CUDA-D2-SAME: "-o" "[[GPUBINARY2:[^"]*]]"
// CUDA-D2-SAME: "-x" "cuda"

// Match no device-side compilation
// CUDA-ND-NOT: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// CUDA-ND-SAME-NOT: "-fcuda-is-device"

// Match host-side preprocessor job with -save-temps
// CUDA-HS: "-cc1" "-triple" "x86_64--linux-gnu"
// CUDA-HS-SAME: "-aux-triple" "nvptx64-nvidia-cuda"
// CUDA-HS-SAME-NOT: "-fcuda-is-device"
// CUDA-HS-SAME: "-x" "cuda"

// Match host-side compilation
// CUDA-H: "-cc1" "-triple" "x86_64--linux-gnu"
// CUDA-H-SAME: "-aux-triple" "nvptx64-nvidia-cuda"
// CUDA-H-SAME-NOT: "-fcuda-is-device"
// CUDA-H-SAME: "-o" "[[HOSTOUTPUT:[^"]*]]"
// CUDA-HNS-SAME: "-x" "cuda"
// CUDA-HS-SAME: "-x" "cuda-cpp-output"
// CUDA-H-I1-SAME: "-fcuda-include-gpubinary" "[[GPUBINARY1]]"
// CUDA-H-I2-SAME: "-fcuda-include-gpubinary" "[[GPUBINARY2]]"

// Match external assembler that uses compilation output
// CUDA-H-AS: "-o" "{{.*}}.o" "[[HOSTOUTPUT]]"

// Match no GPU code inclusion.
// CUDA-H-NI-NOT: "-fcuda-include-gpubinary"

// Match no CUDA compilation
// CUDA-NH-NOT: "-cc1" "-triple"
// CUDA-NH-SAME-NOT: "-x" "cuda"

// Match linker
// CUDA-L: "{{.*}}{{ld|link}}{{(.exe)?}}"
// CUDA-L-SAME: "[[HOSTOUTPUT]]"

// Match no linker
// CUDA-NL-NOT: "{{.*}}{{ld|link}}{{(.exe)?}}"

// CUDA-NO-UNUSED-CHO-NOT: warning: argument unused during compilation: '--cuda-host-only'
// CUDA-UNUSED-CDO: warning: argument unused during compilation: '--cuda-device-only'
// CUDA-NO-UNUSED-CDO-NOT: warning: argument unused during compilation: '--cuda-device-only'
